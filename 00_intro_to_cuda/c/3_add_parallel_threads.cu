/**********************************************************************\
 * Author: Jose A. Iglesias-Guitian                                   *
 * C/C++ code							      *
 * Introduction to CUDA						      *
/**********************************************************************/

// Instructions: How to compile this program.
// nvcc 3_add_parallel_threads.cu -L /usr/local/cuda/lib -lcudart -o 3_add_parallel_threads

// Multiple blocks, one thread each


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c, int N)  {
    int id = blockIdx.x*blockDim.x + threadIdx.x;

    if( id < N ){
        c[id] = a[id] + b[id];
    }
}

int main() {
   // Vector size
   int N = 100000;
 
   // Host vectors
   int *a, *b;
   int *c; // output vector
 
   // Device vectors
   int *d_a, *d_b;
   int *d_c;  // device copies
 
   // Size in bytes of each vector
   size_t size = N*sizeof(int);
 
   // Allocate host memory
   a = (int*)malloc(size);
   b = (int*)malloc(size);
   c = (int*)malloc(size);
 
   // Allocate device memory
   hipMalloc((void **) &d_a, size);
   hipMalloc((void **) &d_b, size);
   hipMalloc((void **) &d_c, size);
 
   // Initialize host vectors
   for( int i = 0; i < N; i++) {
     a[i] = i;
     b[i] = -(i-1);
   }
 
   // Copy host input vectors to device
   hipMemcpy( d_a, a, size, hipMemcpyHostToDevice );
   hipMemcpy( d_b, b, size, hipMemcpyHostToDevice );
 
   // Number of thread per block
   int threadCount = 128;
 
   // Number of blocks per grid
   int blockCount = (int)ceil((float)N/threadCount);
 
   // Launch add() on GPU
   add<<<blockCount,threadCount>>>(d_a, d_b, d_c, N);
 
   // Copy result to host
   hipMemcpy( c, d_c, size, hipMemcpyDeviceToHost);
 
   // Results should sum up to N
   int sum = 0;
   for (int i = 0; i < N; i++) {
     if (i < 5) {
       printf("%d + %d = %d\n", a[i], b[i], c[i]);
     }
     sum += c[i];
   }
   printf("...\n");
 
   printf("Should be %d\nResults: %d\n", N,sum);
 
   // Cleanup host
   free(a);
   free(b);
   free(c);
 
   // Cleanup device
   hipFree(d_a);
   hipFree(d_b);
   hipFree(d_c);
 
   return 0;
}
