/**********************************************************************\
 * Author: Jose A. Iglesias-Guitian                                   *
 * C/C++ code							      *
 * Introduction to CUDA						      *
/**********************************************************************/

// Instructions: How to compile this program.
// nvcc 1_add.cu -L /usr/local/cuda/lib -lcudart -o 1_add


#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void add(int *a, int *b, int *c)
{
    printf("(GPU) Hello from thread id (%d,%d)\n", threadIdx.x, threadIdx.y);
    *c = *a + *b;
}

int main(void) {
    int a, b, c;	            // host copies of a, b, c
    int *d_a, *d_b, *d_c;	     // device copies of a, b, c
    int size = sizeof(int);
    // Allocate space for device copies of a, b, c
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);
    hipMalloc((void **)&d_c, size);
    // Setup input values
    a = 2;
    b = 7;
    // Copy inputs to device
    hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

    // Launch add() kernel on GPU
    add<<<1,1>>>(d_a, d_b, d_c);

    // Copy result back to host
    hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
    printf("(CPU) Add result is %d\n", c);
    // Cleanup
    hipFree(d_a); hipFree(d_b); hipFree(d_c);
    return 0;
}
